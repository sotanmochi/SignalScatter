#include "hip/hip_runtime.h"
#include "BitonicSortSample.h"
#include "../../../src/cpp/Point.h"
#include "../../../src/cuda/BitonicSort.h"

#include <cfloat>
#include <chrono>
#include <iostream>
#include <random>

#define BLOCK_SIZE 32 // ToDo

SignalScatter::BitonicSortSample::BitonicSortSample(uint32_t size)
{
    std::random_device seed_gen;
    std::default_random_engine engine(seed_gen());
    std::uniform_real_distribution<float> dist(-256, 256);

    _size = size;

    _distanceMatrix = new float[size * size];
    _distanceMatrixMemorySize = size * size * sizeof(float);

    _packedIdMatrix = new uint32_t[size * size];
    _packedIdMatrixMemorySize = size * size * sizeof(uint32_t);

    _points = new Point[size];
    _pointListMemorySize = size * sizeof(Point);

    for (int i = 0; i < size; i++)
    {
        _points[i].Id = i;
        _points[i].PositionX = dist(engine);
        _points[i].PositionY = dist(engine);
        _points[i].PositionZ = dist(engine);
    }

    std::cout << "--------------------" << std::endl;
    std::cout << "Bitonic Sort Sample" << std::endl;
    std::cout << "--------------------" << std::endl;
    std::cout << "----------" << std::endl;
    std::cout << "N: " << _size << std::endl;
    std::cout << "N x N: " << (_size * _size) << std::endl;
    std::cout << "PointListMemorySize: " << _pointListMemorySize << " [Bytes]" << std::endl;
    std::cout << "DistanceMatrixMemorySize: " << _distanceMatrixMemorySize << " [Bytes]" << std::endl;
    std::cout << "PackedIdMatrixMemorySize: " << _packedIdMatrixMemorySize << " [Bytes]" << std::endl;
    std::cout << "----------" << std::endl;

    hipError_t err;

	err = hipMalloc((void **)&_d_PointList, _pointListMemorySize);
    std::cout << "CudaMalloc: " << err << std::endl;

	err = hipMalloc((void **)&_d_DistanceMatrix, _distanceMatrixMemorySize);
    std::cout << "CudaMalloc: " << err << std::endl;

	err = hipMalloc((void **)&_d_PackedIdMatrix, _packedIdMatrixMemorySize);
    std::cout << "CudaMalloc: " << err << std::endl;

	err = hipMalloc((void **)&_d_DistanceMatrixOut, _distanceMatrixMemorySize);
    std::cout << "CudaMalloc: " << err << std::endl;

	err = hipMalloc((void **)&_d_PackedIdMatrixOut, _packedIdMatrixMemorySize);
    std::cout << "CudaMalloc: " << err << std::endl;
}

SignalScatter::BitonicSortSample::~BitonicSortSample()
{
    hipError_t err;

    err = hipFree(_d_PointList);
    std::cout << "CudaFree: " << err << std::endl;

    err = hipFree(_d_DistanceMatrix);
    std::cout << "CudaFree: " << err << std::endl;

    err = hipFree(_d_PackedIdMatrix);
    std::cout << "CudaFree: " << err << std::endl;

    err = hipFree(_d_DistanceMatrixOut);
    std::cout << "CudaFree: " << err << std::endl;

    err = hipFree(_d_PackedIdMatrixOut);
    std::cout << "CudaFree: " << err << std::endl;
}

__global__ void CalculateDistanceKernel(int n, SignalScatter::Point *points, 
                                        float *distanceMatrix, uint32_t *packedIdMatrix)
{
	unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;

	unsigned int a_index = row;
	unsigned int b_index = col;
	unsigned int c_index = row * n + col;

	if (row >= n || col >= n)
	{
		return;
	}

    float dx = points[a_index].PositionX - points[b_index].PositionX;
    float dy = points[a_index].PositionY - points[b_index].PositionY;
    float dz = points[a_index].PositionZ - points[b_index].PositionZ;

	distanceMatrix[c_index] = dx * dx + dy * dy + dz * dz; // Squared Distance
    packedIdMatrix[c_index] = (uint)((a_index & 0xFFFF) << 16 | b_index);
}

void CalculateDistance(int n, SignalScatter::Point *points, 
                       float *distanceMatrix, uint32_t *packedIdMatrix)
{
	int blockWidth  = BLOCK_SIZE;
	int blockHeight = BLOCK_SIZE;
	int gridWidth   = ceil((float)n/blockWidth);
	int gridHeight  = ceil((float)n/blockHeight);

    std::cout << "----------" << std::endl;
    std::cout << "gridWidth: "  << gridWidth << std::endl;
    std::cout << "gridHeight: " << gridHeight << std::endl;
    std::cout << "blockWidth: "  << blockWidth << std::endl;
    std::cout << "blockHeight: " << blockHeight << std::endl;
    std::cout << "----------" << std::endl;

	dim3 blockSize(blockWidth, blockHeight, 1);
	dim3 gridSize(gridWidth, gridHeight, 1);

    CalculateDistanceKernel<<<gridSize, blockSize>>>(n, points, distanceMatrix, packedIdMatrix);

	hipError_t err = hipGetLastError();

    std::cout << "----------" << std::endl;
    std::cout << "CalculateDistance: " << err << std::endl;
    std::cout << "----------" << std::endl;
}

void PrintDistanceMatrix(std::string title, uint32_t size, float *distanceMatrix, uint32_t *packedIdMatrix)
{
    std::cout << "-----" << std::endl;
    std::cout << title << std::endl;
    std::cout << "-----" << std::endl;
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            int pi = packedIdMatrix[i*size + j] & 0xFFFF0000 >> 16;
            int pj = packedIdMatrix[i*size + j] & 0xFFFF;
            std::cout << "D[" << i << "][" << j << "]: " << distanceMatrix[i*size + j] << " (" << pi << ", " << pj << ")" << std::endl;
        }
        std::cout << "-----" << std::endl;
    }
    std::cout << "-----" << std::endl;
}

void SignalScatter::BitonicSortSample::Run()
{
    hipError_t err;
    std::chrono::system_clock::time_point start, end;
    double elapsedTimeMilliseconds;

    std::cout << "======================================================================" << std::endl;
    std::cout << std::endl;
    std::cout << "----------" << std::endl;
    std::cout << "Distance Calculation on Accelerator (using CUDA)" << std::endl;
    std::cout << "----------" << std::endl;

	err = hipMemcpy(_d_PointList, _points, _pointListMemorySize, hipMemcpyHostToDevice);
    std::cout << "CudaMemcpyHostToDevice: " << err << std::endl;

    start = std::chrono::system_clock::now();

    std::cout << "Start CalculateDistance" << std::endl;
    CalculateDistance(_size, _d_PointList, _d_DistanceMatrix, _d_PackedIdMatrix);

    end = std::chrono::system_clock::now();
    elapsedTimeMilliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "----------" << std::endl;
    std::cout << "Elapsed time: " << elapsedTimeMilliseconds << " [ms]" << std::endl;
    std::cout << "----------" << std::endl;

    std::cout << "----------" << std::endl;
    std::cout << "Bitonic Sort on Accelerator (using CUDA)" << std::endl;
    std::cout << "----------" << std::endl;

	err = hipMemcpy(_distanceMatrix, _d_DistanceMatrix, _distanceMatrixMemorySize, hipMemcpyDeviceToHost);
    std::cout << "CudaMemcpyDeviceToHost: " << err << std::endl;

	err = hipMemcpy(_packedIdMatrix, _d_PackedIdMatrix, _packedIdMatrixMemorySize, hipMemcpyDeviceToHost);
    std::cout << "CudaMemcpyDeviceToHost: " << err << std::endl;

    PrintDistanceMatrix("Before BitonicSort", _size, _distanceMatrix, _packedIdMatrix);

    start = std::chrono::system_clock::now();

    uint ascending = 1;
    uint batchSize = _size;
    uint arrayLength = _size;

    std::cout << "Start BitonicSort" << std::endl;
    bitonicSort(_d_DistanceMatrixOut, _d_PackedIdMatrixOut, _d_DistanceMatrix, _d_PackedIdMatrix, batchSize, arrayLength, ascending);

    end = std::chrono::system_clock::now();
    elapsedTimeMilliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "----------" << std::endl;
    std::cout << "Elapsed time: " << elapsedTimeMilliseconds << " [ms]" << std::endl;
    std::cout << "----------" << std::endl;

    std::cout << "----------" << std::endl;
    std::cout << "Memory Copy (Device -> Host)" << std::endl;
    std::cout << "----------" << std::endl;

    start = std::chrono::system_clock::now();

	err = hipMemcpy(_distanceMatrix, _d_DistanceMatrixOut, _distanceMatrixMemorySize, hipMemcpyDeviceToHost);
    std::cout << "CudaMemcpyDeviceToHost: " << err << std::endl;

	err = hipMemcpy(_packedIdMatrix, _d_PackedIdMatrixOut, _packedIdMatrixMemorySize, hipMemcpyDeviceToHost);
    std::cout << "CudaMemcpyDeviceToHost: " << err << std::endl;

    end = std::chrono::system_clock::now();
    elapsedTimeMilliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "----------" << std::endl;
    std::cout << "Elapsed time: " << elapsedTimeMilliseconds << " [ms]" << std::endl;
    std::cout << "----------" << std::endl;

    PrintDistanceMatrix("After BitonicSort", _size, _distanceMatrix, _packedIdMatrix);

    std::cout << "======================================================================" << std::endl;
}